#include "hip/hip_runtime.h"
#include "matrix.hpp"

// constructor for a multidim container
matrix::matrix(int x, int y) {
    dim_x = x;
    dim_y = y;

    dev_alloc = false;
    host_alloc = false;

}

// Constructor for a general container
matrix::matrix() {
    dim_x = 1;
    dim_y = 1;

    dev_alloc = false;
    host_alloc = false;

}

// allocates memory on GPU if not allocated
void matrix::alloc_dev_mem() {
    if(!dev_alloc) {
        dev_data = nullptr;
        hipMalloc(&dev_data, dim_x * dim_y * sizeof(float));
        dev_alloc = true;
    }
}

// allocates memory on Host if not allocated
void matrix::alloc_host_mem() {
    if(!host_alloc) {
        host_data = new float[dim_x * dim_y];
        host_alloc = true;
    }
}

// function to allocate memory on both host and device
void matrix::mem_alloc() {
    alloc_host_mem();
    alloc_dev_mem();
}

// function to move memory from the host on to the device
void matrix::copy_host_to_dev() {
    if(dev_alloc && host_alloc) {
        hipMemcpy(dev_data, host_data, dim_x * dim_y *sizeof(float), hipMemcpyHostToDevice);
    }
}

// function to move device memory back to host
void matrix::copy_dev_to_host() {
    if(dev_alloc && host_alloc) {
        hipMemcpy(host_data, dev_data, dim_x * dim_y *sizeof(float), hipMemcpyDeviceToHost);
    }
}
// function will clear the device and host memory set both with a new array
void matrix::set_mem(float* new_vals, int x, int y) {
    if(host_alloc) {
        delete host_data;
        host_alloc = false;
    }

    if(dev_alloc) {
        hipFree(dev_data);
        dev_alloc = false;
    }

    dim_x = x;
    dim_y = y;
    mem_alloc();
    memcpy(host_data, new_vals, (x*y));
    copy_host_to_dev();
}

// function to update memory if the memory is the same size.
// THIS FUNCTION WILL NOT CHECK MEMORY SIZE must check before.
void matrix::update_mem(float* new_vals) {
    memcpy(host_data, new_vals, dim_x * dim_y);
    copy_host_to_dev();
}

matrix::~matrix() {
    delete host_data;
    hipFree(dev_data);
}

void matrix::print() {
    for(int i = 0; i < dim_x*dim_y; i++) {
        printf("%f ", host_data[i]);
    }
    printf("\n");
}

//===== ALL FUNCTIONS AFTER FOR TESTING ====
void matrix::pst_vals() {
    for(int i = 0; i < dim_x * dim_y; i++) {
        host_data[i] = 1;
    }
}
__global__ void add_one_vec(float* dev_mem, int size) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    printf("Size: %d\n", size);
    printf("blockDimx: %d\n", blockDim.x);
    printf("blockIdxx: %d\n", blockIdx.x);
    printf("threadIdxx: %d\n", threadIdx.x);
    //int lcnt = 0;
    printf("TID: %d\n", tid);
    dev_mem[tid] += 1;
}

void matrix::add_one() {

    dim3 block_size(2,2);
    dim3 num_blocks( 32, 32);

    //add_one_vec<<<num_blocks, block_size>>>(dev_data, dim_x*dim_y);

    add_one_vec<<<32,32>>>(dev_data, dim_x*dim_y);
}




