#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrix.hpp"

#define T 128

__global__ void _mat_mul(float *mat1, float *mat2, float*result, int c1, int c2) {
    __shared__ float shared[T];
    int tid = threadIdx.x;
     
    if(tid < c1)
        shared[tid] = mat1[blockIdx.x / c2 * c1 + tid] * mat2[blockIdx.x % c2 + c2 * tid];
    else
        shared[tid] = 0;        
    
    __syncthreads();

    for(int s = 1; s < blockDim.x; s*=2) {
        if(tid % (2 * s) == 0) {
            shared[tid] = shared[tid] + shared[tid + s];
        }

        __syncthreads();
    }

    if(tid == 0) {
        result[blockIdx.x] = shared[0];
    }
}

void mat_mul(matrix *mat1, matrix *mat2, matrix *result) {
    if(mat1->on_device && mat2->on_device && result->on_device) {
        int r1 = mat1->num_rows, c1 = mat1->num_cols;
        int r2 = mat2->num_rows, c2 = mat2->num_cols;
        if(c1 == r2)
            _mat_mul<<<r1 * c2, T>>>(mat1->device_data, mat2->device_data, result->device_data, c1, c2);
        else
            printf("Incompatible matrix dimensions. m1 is %d x %d, m2 is %d x %d\n", r1, c1, r2, c2);
    }
    else
        printf("make sure input matrices and output matrix have been moved to device");
}
